﻿/* Copyright © 2017-2020 ABBYY Production LLC

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

	http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
--------------------------------------------------------------------------------------------------------------*/

#include <NeoMathEngine/NeoMathEngineDefs.h>

#ifdef NEOML_USE_CUDA

#include <CudaMathEngine.h>
#include <CudaDevice.h>
#include <CudaCommon.h>
#include <MathEngineCommon.h>
#include <MathEngineDnnDropout.h>
#include <MemoryHandleInternal.h>

#include <Kernels/CudaDnnDropoutKernels.h>

namespace NeoML {

void CCudaMathEngine::Dropout( const CDropoutDesc& dropoutDesc, const CFloatHandle& inputData, const CFloatHandle& outputData )
{
	ASSERT_EXPR( inputData.GetMathEngine() == this );
	ASSERT_EXPR( outputData.GetMathEngine() == this );
	SetCudaDevice( device->DeviceNumber );

	const CMathEngineDropoutDesc& desc = static_cast<const CMathEngineDropoutDesc&>( dropoutDesc );
	const CBlobDesc& input = desc.Input;

	if( desc.ForwardRate == 1.f ) {
		VectorCopy( outputData, inputData, input.BlobSize() );
		return;
	}

	const int objectSize = desc.IsSpatial ? input.Channels() : input.ObjectSize();
	const int batchLength = desc.IsBatchwise ? input.ObjectCount() : input.BatchLength();
	const int batchWidth = input.ObjectCount() / batchLength;
	const int maskSize = batchWidth * objectSize;

	ASSERT_EXPR( desc.Mask.Size() == maskSize );

	if( !desc.IsSpatial ) {
		MultiplyMatrixByDiagMatrix( inputData, batchLength, maskSize, desc.Mask.GetHandle(), outputData, desc.Output.BlobSize() );
		return;
	}

	dim3 blockCount;
	dim3 threadCount;

	getCudaTaskGrid3D( blockCount, threadCount, input.ObjectCount(), input.ObjectSize() / objectSize,
		objectSize );
	ChannelLastBlobSpatialDropoutKernel<<<blockCount, threadCount>>>( GetRaw( inputData ),
		GetRaw( desc.Mask.GetHandle() ), GetRaw( outputData ), input.ObjectCount(), input.ObjectSize(),
		batchWidth, objectSize );
}

CDropoutDesc* CCudaMathEngine::InitDropout( float rate, bool isSpatial, bool isBatchwise,
	const CBlobDesc& input, const CBlobDesc& output, int seed )
{
	return new CMathEngineDropoutDesc( mathEngine(), rate, isSpatial, isBatchwise, input, output, seed );
}

} // namespace NeoML

#endif // NEOML_USE_CUDA
