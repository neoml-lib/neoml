#include "hip/hip_runtime.h"
/* Copyright © 2017-2020 ABBYY Production LLC

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

	http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
--------------------------------------------------------------------------------------------------------------*/

#include <NeoMathEngine/NeoMathEngineDefs.h>

#ifdef NEOML_USE_CUDA

#include <CudaMathEngine.h>
#include <CudaMathEngineDnnConvs.h>
#include <MemoryHandleInternal.h>
#include <MathEngineCommon.h>
#include <CudaDevice.h>
#include <CudaCommon.h>

#include <Kernels/CudaDnnTimeConvKernels.h>

namespace NeoML {

CTimeConvolutionDesc* CCudaMathEngine::InitTimeConvolution( const CBlobDesc& source,
	int stride, int paddingFront, int paddingBack, int dilation,
	const CBlobDesc& filter, const CBlobDesc& result )
{
	ASSERT_EXPR( stride > 0 );
	ASSERT_EXPR( paddingFront >= 0 );
	ASSERT_EXPR( paddingBack >= 0 );
	ASSERT_EXPR( dilation > 0 );
	ASSERT_EXPR( filter.BatchLength() == 1 );
	ASSERT_EXPR( filter.Width() == 1 );
	ASSERT_EXPR( filter.Depth() == 1 );
	ASSERT_EXPR( filter.Channels() == source.ObjectSize() );
	ASSERT_EXPR( source.BatchLength() + paddingFront + paddingBack >= ( filter.Height() - 1 ) * dilation + 1 );
	ASSERT_EXPR( result.BatchLength() == ( source.BatchLength() - ( filter.Height() - 1 ) * dilation - 1 + paddingFront + paddingBack ) / stride + 1 );
	ASSERT_EXPR( result.BatchWidth() == source.BatchWidth() );
	ASSERT_EXPR( result.ListSize() == 1 && source.ListSize() == 1 );
	ASSERT_EXPR( result.Width() == 1 );
	ASSERT_EXPR( result.Height() == 1 );
	ASSERT_EXPR( result.Depth() == 1 );
	ASSERT_EXPR( result.Channels() == filter.BatchWidth() );
	ASSERT_EXPR( paddingFront < ( filter.Height() - 1 ) * dilation + 1 );
	ASSERT_EXPR( paddingBack < ( filter.Height() - 1 ) * dilation + 1 );

	CCudaTimeConvolutionDesc* desc = new CCudaTimeConvolutionDesc();
	desc->Internal.Source = source;
	desc->Internal.Filter = filter;
	desc->Internal.Result = result;
	desc->Internal.Stride = stride;
	desc->Internal.PaddingFront = paddingFront;
	desc->Internal.PaddingBack = paddingBack;
	desc->Internal.Dilation = dilation;
	return desc;
}

void CCudaMathEngine::BlobTimeConvolution( const CTimeConvolutionDesc& convDesc,
	const CFloatHandle& sourceData, const CFloatHandle& filterData, const CFloatHandle& freeTermData,
	const CFloatHandle& resultData )
{
	ASSERT_EXPR( sourceData.GetMathEngine() == this );
	ASSERT_EXPR( filterData.GetMathEngine() == this );
	ASSERT_EXPR( freeTermData.GetMathEngine() == this );
	ASSERT_EXPR( resultData.GetMathEngine() == this );

	const CCudaTimeConvolutionDescInternal& desc = static_cast<const CCudaTimeConvolutionDesc&>( convDesc ).Internal;
	const CCudaBlobDesc& source = desc.Source;
	const CCudaBlobDesc& filter = desc.Filter;
	const CCudaBlobDesc& result = desc.Result;

	if( filter.Height() == 1 && desc.Stride == 1 ) {
		// This assert has already been checked in InitTimeConvolution
		ASSERT_EXPR( desc.PaddingFront == 0 && desc.PaddingBack == 0 );
		// Trivial case
		MultiplyMatrixByTransposedMatrix(sourceData,
			source.BatchLength() * source.BatchWidth(), source.ObjectSize(), source.ObjectSize(),
			filterData, filter.ObjectCount(), source.ObjectSize(),
			resultData + desc.PaddingFront * filter.ObjectCount(), filter.ObjectCount(), result.BlobSize());
	} else {
		// Convolution through temp matrix
		const int tempMatrixWidth = filter.ObjectSize();
		const int tempMatrixHeight = result.BlobSize() / filter.ObjectCount();
		// Max amount of memory allowed is a half of math engine's free memory
		const int maxInMemoryHeight = max( 1,
			min( static_cast<int>( GetFreeMemorySize() / 2 / ( sizeof( float ) * tempMatrixWidth ) ), tempMatrixHeight ) );

		int matrixRowIndex = 0;
		CFloatHandle currResult = resultData;
		CFloatHandleStackVar tempMatrixPart( mathEngine(), maxInMemoryHeight * tempMatrixWidth );

		// Build temp matrix part by part and add filterDiff of that part
		while( matrixRowIndex < tempMatrixHeight ) {
			const int currPartHeight = min( tempMatrixHeight - matrixRowIndex, maxInMemoryHeight );

			dim3 blockCount;
			dim3 threadCount;
			getCudaTaskGrid2D( blockCount, threadCount, currPartHeight, tempMatrixWidth );

			BuildTempMatrixKernel<<<blockCount, threadCount>>>( desc, GetRaw( sourceData ), currPartHeight,
				tempMatrixWidth, GetRaw( tempMatrixPart.GetHandle() ), matrixRowIndex );
			MultiplyMatrixByTransposedMatrix(tempMatrixPart, currPartHeight, tempMatrixWidth, tempMatrixWidth,
				filterData, filter.ObjectCount(), tempMatrixWidth,
				currResult, filter.ObjectCount(), result.BlobSize());

			matrixRowIndex += currPartHeight;
			currResult += currPartHeight * filter.ObjectCount();
		}
	}

	// Free term
	AddVectorToMatrixRows( 1, resultData, resultData, result.ObjectCount(), result.ObjectSize(), freeTermData );
}

void CCudaMathEngine::BlobTimeConvolutionBackward( const CTimeConvolutionDesc& convDesc,
	const CFloatHandle& outputDiffData, const CFloatHandle& filterData, const CFloatHandle& /*freeTerm*/,
	const CFloatHandle& inputDiffData )
{
	ASSERT_EXPR( outputDiffData.GetMathEngine() == this );
	ASSERT_EXPR( filterData.GetMathEngine() == this );
	ASSERT_EXPR( inputDiffData.GetMathEngine() == this );
	SetCudaDevice( device->DeviceNumber );

	const CCudaTimeConvolutionDescInternal& desc = static_cast<const CCudaTimeConvolutionDesc&>( convDesc ).Internal;
	const CCudaBlobDesc& inputDiff = desc.Source;
	const CCudaBlobDesc& filter = desc.Filter;
	const CCudaBlobDesc& outputDiff = desc.Result;

	if( desc.Stride == 1 && filter.Height() == 1 ) {
		// This assert has already been checked in InitTimeConvolution
		ASSERT_EXPR( desc.PaddingFront == 0 && desc.PaddingBack == 0 );
		// Trivial case
		MultiplyMatrixByMatrix( 1, outputDiffData, outputDiff.ObjectCount(), outputDiff.ObjectSize(),
			filterData, filter.ObjectSize(), inputDiffData, inputDiff.BlobSize() );
	} else {
		// Let's try to build temp matrix
		const int tempMatrixWidth = filter.ObjectSize();
		const int tempMatrixHeight = outputDiff.BlobSize() / filter.ObjectCount();
		// Max amount of memory allowed is a half of math engine's free memory
		const int maxInMemoryHeight = max( 1,
			min( static_cast<int>( GetFreeMemorySize() / 2 / ( sizeof( float ) * tempMatrixWidth ) ), tempMatrixHeight ) );

		int matrixRowIndex = 0;
		CFloatHandle currOutputDiff = outputDiffData;
		CFloatHandleStackVar tempMatrixPart( mathEngine(), maxInMemoryHeight * tempMatrixWidth );

		VectorFill( inputDiffData, 0.f, inputDiff.BlobSize() );
		
		const int combineCount = max( 1, BlobTimeConvolutionBackwardUnpackCombine / filter.Height() );
		const int xSizeNorm = (inputDiff.ObjectSize() + combineCount - 1) / combineCount;
		while( matrixRowIndex < tempMatrixHeight ) {
			const int currPartHeight = min( tempMatrixHeight - matrixRowIndex, maxInMemoryHeight );

			MultiplyMatrixByMatrix( 1, currOutputDiff, currPartHeight, outputDiff.ObjectSize(),
				filterData, filter.ObjectSize(), tempMatrixPart, maxInMemoryHeight * tempMatrixWidth );

			dim3 blockCount;
			dim3 threadCount;
			getCudaTaskGrid2DMinYX(1, 512, blockCount, threadCount, inputDiff.ObjectCount(), xSizeNorm);
			BlobTimeConvolutionBackwardUnpackKernel<<<blockCount, threadCount>>>( desc, GetRaw( filterData ),
				GetRaw( inputDiffData ), xSizeNorm, combineCount, GetRaw( tempMatrixPart.GetHandle() ), matrixRowIndex, currPartHeight );

			currOutputDiff += currPartHeight * outputDiff.ObjectSize();
			matrixRowIndex += currPartHeight;
		}
	}
}

void CCudaMathEngine::BlobTimeConvolutionLearnAdd( const CTimeConvolutionDesc& convDesc, const CFloatHandle& inputData,
	const CFloatHandle& outputDiffData, const CFloatHandle& filterDiffData, const CFloatHandle& freeTermDiffData )
{
	ASSERT_EXPR( inputData.GetMathEngine() == this );
	ASSERT_EXPR( outputDiffData.GetMathEngine() == this );
	ASSERT_EXPR( filterDiffData.GetMathEngine() == this );
	ASSERT_EXPR( freeTermDiffData.GetMathEngine() == this );
	SetCudaDevice( device->DeviceNumber );

	const CCudaTimeConvolutionDescInternal& desc = static_cast<const CCudaTimeConvolutionDesc&>( convDesc ).Internal;
	const CCudaBlobDesc& filterDiff = desc.Filter;
	const CCudaBlobDesc& outputDiff = desc.Result;

	// Train the filter
	if( filterDiff.Height() == 1 && desc.Stride == 1 ) {
		// This assert has already been checked in InitTimeConvolution
		ASSERT_EXPR( desc.PaddingFront == 0 && desc.PaddingBack == 0 );
		// Trivial case
		MultiplyTransposedMatrixByMatrixAndAdd( outputDiffData, desc.Source.ObjectCount(),
			outputDiff.ObjectSize(), outputDiff.ObjectSize(), inputData, desc.Source.ObjectSize(),
			desc.Source.ObjectSize(), filterDiffData, filterDiff.ObjectSize(), filterDiff.BlobSize() );
	} else {
		// Let's try to build temp matrix
		const int tempMatrixWidth = filterDiff.ObjectSize();
		const int tempMatrixHeight = outputDiff.BlobSize() / filterDiff.ObjectCount();
		// Max amount of memory allowed is a half of math engine's free memory
		const int maxInMemoryHeight = min( static_cast<int>( GetFreeMemorySize() / 2 / ( sizeof( float ) * tempMatrixWidth ) ),
			tempMatrixHeight );

		if( maxInMemoryHeight == 0 ) {
			// naive implementatino which doesn't use additional memory
			int blockCount;
			int threadCount;
			getCudaTaskGrid( blockCount, threadCount, desc.Filter.BlobSize() );
			BlobTimeConvolutionLearnFilterKernel<<<blockCount, threadCount>>>( desc, GetRaw( inputData ),
				GetRaw( outputDiffData ), GetRaw( filterDiffData ) );
		} else {
			int matrixRowIndex = 0;
			CFloatHandle currOutputDiff = outputDiffData;
			CFloatHandleStackVar tempMatrixPart( mathEngine(), maxInMemoryHeight * tempMatrixWidth );
			const int filterCount = desc.Result.ObjectSize();

			// Build temp matrix part by part and add filterDiff of that part
			while( matrixRowIndex < tempMatrixHeight ) {
				const int currPartHeight = min( tempMatrixHeight - matrixRowIndex, maxInMemoryHeight );

				dim3 blockCount;
				dim3 threadCount;
				getCudaTaskGrid2D( blockCount, threadCount, currPartHeight, tempMatrixWidth );

				BuildTempMatrixKernel<<<blockCount, threadCount>>>( desc, GetRaw( inputData ), currPartHeight,
					tempMatrixWidth, GetRaw( tempMatrixPart.GetHandle() ), matrixRowIndex );
				MultiplyTransposedMatrixByMatrixAndAdd( currOutputDiff, currPartHeight, filterCount, filterCount,
					tempMatrixPart.GetHandle(), tempMatrixWidth, tempMatrixWidth, filterDiffData, tempMatrixWidth, filterDiff.BlobSize() );

				matrixRowIndex += currPartHeight;
				currOutputDiff += currPartHeight * filterCount;
			}
		}
	}

	// Train the free term
	SumMatrixRowsAdd( 1, freeTermDiffData, outputDiffData, outputDiff.ObjectCount(), filterDiff.ObjectCount() );
}

} // namespace NeoML

#endif // NEOML_USE_CUDA
