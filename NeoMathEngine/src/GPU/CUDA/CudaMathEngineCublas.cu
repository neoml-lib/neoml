﻿/* Copyright © 2017-2020 ABBYY Production LLC

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

	http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
--------------------------------------------------------------------------------------------------------------*/

#include <NeoMathEngine/NeoMathEngineDefs.h>

#ifdef NEOML_USE_CUDA

#include <CudaMathEngine.h>
#include <CudaCommon.h>
#include <CudaAssert.h>
#include <CublasFunctions.h>
#include <MathEngineCommon.h>
#include <MemoryHandleInternal.h>

#include <hip/hip_runtime_api.h>

namespace NeoML {

void CCudaMathEngine::VectorDotProduct(const CConstFloatHandle& firstHandle, const CConstFloatHandle& secondHandle,
    int vectorSize, const CFloatHandle& resultHandle)
{
    ASSERT_EXPR( firstHandle.GetMathEngine() == this );
    ASSERT_EXPR( secondHandle.GetMathEngine() == this );
    ASSERT_EXPR( resultHandle.GetMathEngine() == this );

	ASSERT_CUBLAS( cublas->Sdot( cublasHandle, vectorSize, GetRaw( firstHandle ), 1,
		GetRaw( secondHandle ), 1, GetRaw( resultHandle ) ) );
}

void CCudaMathEngine::VectorMultiplyAndAdd( const CConstFloatHandle& firstHandle, const CConstFloatHandle& secondHandle,
	const CFloatHandle& resultHandle, int vectorSize, const CConstFloatHandle& multHandle )
{
	ASSERT_EXPR( firstHandle.GetMathEngine() == this );
	ASSERT_EXPR( secondHandle.GetMathEngine() == this );
	ASSERT_EXPR( resultHandle.GetMathEngine() == this );
	ASSERT_EXPR( multHandle.GetMathEngine() == this );

	const float* first = GetRaw( firstHandle );
	const float* second = GetRaw( secondHandle );
	float* result = GetRaw( resultHandle );
	const float* mult = GetRaw( multHandle );

	if( result != first ) {
		ASSERT_CUDA( hipMemcpy( result, first, vectorSize * sizeof( float ), hipMemcpyDeviceToDevice ) );
	}
	ASSERT_CUBLAS( cublas->Saxpy( cublasHandle, vectorSize, mult, second, 1, result, 1 ) );
}

void CCudaMathEngine::MultiplyMatrixByTransposedMatrix( const CConstFloatHandle& firstHandle, int firstHeight,
	int firstWidth, int firstRowSize, const CConstFloatHandle& secondHandle, int secondHeight, int secondRowSize,
	const CFloatHandle& resultHandle, int resultRowSize, int )
{
	ASSERT_EXPR( firstHandle.GetMathEngine() == this );
	ASSERT_EXPR( secondHandle.GetMathEngine() == this );
	ASSERT_EXPR( resultHandle.GetMathEngine() == this );

	ASSERT_CUBLAS( cublas->Sgemm( cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, secondHeight, firstHeight, firstWidth,
		CCudaConst::One, GetRaw( secondHandle ), secondRowSize, GetRaw( firstHandle ), firstRowSize, CCudaConst::Zero,
		GetRaw( resultHandle ), resultRowSize ) );
}

void CCudaMathEngine::MultiplyMatrixByTransposedMatrix( int batchSize, const CConstFloatHandle& firstHandle,
	int firstHeight, int firstWidth, const CConstFloatHandle& secondHandle, int secondHeight,
	const CFloatHandle& resultHandle, int resultBufferSize )
{
	ASSERT_EXPR( firstHandle.GetMathEngine() == this );
	ASSERT_EXPR( secondHandle.GetMathEngine() == this );
	ASSERT_EXPR( resultHandle.GetMathEngine() == this );

	ASSERT_CUBLAS( cublas->SgemmStridedBatched( cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, secondHeight,
		firstHeight, firstWidth, CCudaConst::One, GetRaw( secondHandle ), firstWidth, firstWidth * secondHeight,
		GetRaw( firstHandle ), firstWidth, firstHeight * firstWidth, CCudaConst::Zero, GetRaw( resultHandle ),
		secondHeight, secondHeight * firstHeight, batchSize ) );
}

void CCudaMathEngine::MultiplyTransposedMatrixByMatrixAndAdd( const CConstFloatHandle& firstHandle, int firstHeight,
	int firstWidth, int firstRowSize, const CConstFloatHandle& secondHandle, int secondWidth, int secondRowSize,
	const CFloatHandle& resultHandle, int resultRowSize, int )
{
	ASSERT_EXPR( firstHandle.GetMathEngine() == this );
	ASSERT_EXPR( secondHandle.GetMathEngine() == this );
	ASSERT_EXPR( resultHandle.GetMathEngine() == this );

	ASSERT_CUBLAS( cublas->Sgemm( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, secondWidth, firstWidth, firstHeight,
		CCudaConst::One, GetRaw( secondHandle ), secondRowSize, GetRaw( firstHandle ), firstRowSize, CCudaConst::One,
		GetRaw( resultHandle ), resultRowSize ) );
}

void CCudaMathEngine::MultiplyTransposedMatrixByMatrix( int batchSize, const CConstFloatHandle& firstHandle, int firstHeight,
	int firstWidth, const CConstFloatHandle& secondHandle, int secondWidth, const CFloatHandle& resultHandle, int )
{
	ASSERT_EXPR( firstHandle.GetMathEngine() == this );
	ASSERT_EXPR( secondHandle.GetMathEngine() == this );
	ASSERT_EXPR( resultHandle.GetMathEngine() == this );

	ASSERT_CUBLAS( cublas->SgemmStridedBatched( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, secondWidth, firstWidth,
		firstHeight, CCudaConst::One, GetRaw(secondHandle), secondWidth, firstHeight * secondWidth, GetRaw(firstHandle),
		firstWidth, firstHeight * firstWidth, CCudaConst::Zero, GetRaw(resultHandle), secondWidth, firstWidth * secondWidth,
		batchSize ) );
}

void CCudaMathEngine::MultiplyMatrixByMatrix( int batchSize, const CConstFloatHandle& firstHandle, int firstHeight,
	int firstWidth, const CConstFloatHandle& secondHandle, int secondWidth,
	const CFloatHandle& resultHandle, int )
{
	ASSERT_EXPR( firstHandle.GetMathEngine() == this );
	ASSERT_EXPR( secondHandle.GetMathEngine() == this );
	ASSERT_EXPR( resultHandle.GetMathEngine() == this );

	if( batchSize == 1 ) {
		ASSERT_CUBLAS( cublas->Sgemm( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, secondWidth, firstHeight, firstWidth,
			CCudaConst::One, GetRaw( secondHandle ), secondWidth, GetRaw( firstHandle ), firstWidth, CCudaConst::Zero,
			GetRaw( resultHandle ), secondWidth ) );
	} else {
		ASSERT_CUBLAS( cublas->SgemmStridedBatched( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, secondWidth, firstHeight, firstWidth,
			CCudaConst::One, GetRaw( secondHandle ), secondWidth, firstWidth * secondWidth, GetRaw( firstHandle ), firstWidth,
			firstHeight * firstWidth, CCudaConst::Zero, GetRaw( resultHandle ), secondWidth, secondWidth * firstHeight, batchSize ) );
	}
}

void CCudaMathEngine::multiplyMatrixByTransposedMatrixAndAdd(const CConstFloatHandle& firstHandle,
	int firstHeight, int firstWidth, int firstRowSize,
	const CConstFloatHandle& secondHandle, int secondHeight, int secondRowSize,
	const CFloatHandle& resultHandle, int resultRowSize)
{
	ASSERT_CUBLAS( cublas->Sgemm( cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, secondHeight, firstHeight, firstWidth,
		CCudaConst::One, GetRaw( secondHandle ), secondRowSize, GetRaw( firstHandle ), firstRowSize, CCudaConst::One,
		GetRaw( resultHandle ), resultRowSize ) );
}

} // namespace NeoML

#endif // NEOML_USE_CUDA
