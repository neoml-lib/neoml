/* Copyright © 2017-2020 ABBYY Production LLC

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

	http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
--------------------------------------------------------------------------------------------------------------*/

#include <NeoMathEngine/NeoMathEngineDefs.h>

#ifdef NEOML_USE_CUDA

#include <NeoMathEngine/NeoMathEngineException.h>
#include <MathEngineDnnLrn.h>
#include <MemoryHandleInternal.h>
#include <CudaMathEngine.h>
#include <CudaDevice.h>
#include <CudaCommon.h>
#include <Kernels/CudaDnnLrnKernels.h>

namespace NeoML {

CLrnDesc* CCudaMathEngine::InitLrn( const CBlobDesc& source, int windowSize, float bias, float alpha, float beta )
{
	return new CMathEngineLrnDesc( source, windowSize, bias, alpha, beta );
}

void CCudaMathEngine::Lrn( const CLrnDesc& lrnDesc, const CConstFloatHandle& input, const CFloatHandle& invSum,
	const CFloatHandle& invSumBeta , const CFloatHandle& output )
{
	ASSERT_EXPR( input.GetMathEngine() == this );
	ASSERT_EXPR( invSum.IsNull() || invSum.GetMathEngine() == this );
	ASSERT_EXPR( invSumBeta.IsNull() || invSumBeta.GetMathEngine() == this );
	ASSERT_EXPR( output.GetMathEngine() == this );
	SetCudaDevice( device->DeviceNumber );

	const CMathEngineLrnDesc& desc = static_cast<const CMathEngineLrnDesc&>( lrnDesc );

	const int vectorSize = desc.Source.Channels();
	const int vectorCount = desc.Source.BlobSize() / vectorSize;

	dim3 blockCount;
	dim3 threadCount;
	getCudaTaskGrid2D( blockCount, threadCount, vectorCount, vectorSize );

	const float* inputPtr = GetRaw( input );
	float* outputPtr = GetRaw( output );
	float* invSumPtr = invSum.IsNull() ? outputPtr : GetRaw( invSum );
	float* invSumBetaPtr = invSumBeta.IsNull() ? outputPtr : GetRaw( invSumBeta );

	LrnKernel<<<blockCount, threadCount>>>( inputPtr, invSumPtr, invSumBetaPtr, outputPtr, vectorCount, vectorSize,
		desc.WindowSize, desc.Bias, desc.Alpha, desc.Beta ); 
}

void CCudaMathEngine::LrnBackward( const CLrnDesc& lrnDesc, const CConstFloatHandle& input, const CConstFloatHandle& output,
		const CConstFloatHandle& outputDiff, const CConstFloatHandle& invSum, const CConstFloatHandle& invSumBeta,
		const CFloatHandle& inputDiff )
{
	ASSERT_EXPR( input.GetMathEngine() == this );
	ASSERT_EXPR( output.GetMathEngine() == this );
	ASSERT_EXPR( outputDiff.GetMathEngine() == this );
	ASSERT_EXPR( invSum.GetMathEngine() == this );
	ASSERT_EXPR( invSumBeta.GetMathEngine() == this );
	ASSERT_EXPR( inputDiff.GetMathEngine() == this );
	SetCudaDevice( device->DeviceNumber );

	const CMathEngineLrnDesc& desc = static_cast<const CMathEngineLrnDesc&>( lrnDesc );

	const int vectorSize = desc.Source.Channels();
	const int vectorCount = desc.Source.BlobSize() / vectorSize;

	dim3 blockCount;
	dim3 threadCount;
	getCudaTaskGrid2D( blockCount, threadCount, vectorCount, vectorSize );

	const float* inputPtr = GetRaw( input );
	const float* outputPtr = GetRaw( output );
	const float* outputDiffPtr = GetRaw( outputDiff );
	const float* invSumPtr = GetRaw( invSum );
	const float* invSumBetaPtr = GetRaw( invSumBeta );
	float* inputDiffPtr = GetRaw( inputDiff );

	LrnBackwardKernel<<<blockCount, threadCount>>>( inputPtr, outputPtr, outputDiffPtr, invSumPtr, invSumBetaPtr,
		inputDiffPtr, vectorCount, vectorSize, desc.WindowSize, desc.Alpha, desc.Beta ); 
}

} // namespace NeoML

#endif // NEOML_USE_CUDA
